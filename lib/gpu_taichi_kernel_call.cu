#include "hip/hip_runtime.h"
#include "gpu_taichi_kernel_call.cuh"
#include "gpu_taichi_aot_kernel.cuh"
#include <unistd.h>

namespace brain_taichi {
    void launch_taichi_gpu_kernel(hipStream_t stream, void **buffers,
                                  const char *opaque, std::size_t opaque_len) {
        // hipDeviceSynchronize();
        // hipStreamCreateWithFlags(&stream, hipStreamDefault);
        hipStreamSynchronize(stream);
        taichi_kernel->set_cuda_stream(stream);

        OpaqueStruct data = parseOpaque(opaque, opaque_len);

        // restruct shape_list, it's a 2d array and the shape of it is (in_num+out_num, the max of dim_count)
        int param_total_num = data.in_num + data.out_num;
        uint32_t shape_list_2d[param_total_num][8];
        for (int i = 0; i < param_total_num; i++) {
            for (int j = 0; j < 8; j++) {
                shape_list_2d[i][j] = data.shape_list[i * 8 + j];
            }
        }

        taichi_kernel->load(data.kernel_aot_path.c_str());

        for (int i = 0; i < data.in_num; i++) {
            push_input(data.type_list[i], buffers[i], data.ndim_list[i], data.size_list[i], shape_list_2d[i]);
        }

        for (int i = 0; i < data.out_num; i++) {
            push_output(data.type_list[i + data.in_num], buffers[i + data.in_num], data.ndim_list[i + data.in_num], data.size_list[i + data.in_num], shape_list_2d[i + data.in_num]);
        }

        // for (int i = 0; i < data.in_num + data.out_num; i++) {
        //     std::cout << "push args: "<< i << " buffer: " << buffers[i] << " ndim: " << data.ndim_list[i] << " size: " << data.size_list[i] << " shape: " << shape_list_2d[i] << std::endl;
        //     push_args(data.type_list[i], buffers[i], data.ndim_list[i], data.size_list[i], shape_list_2d[i]);
        // }

        taichi_kernel->launch();
        taichi_kernel->runtime_.wait();
        taichi_kernel->clear_args();
    }
}
