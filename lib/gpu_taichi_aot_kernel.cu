#include "gpu_taichi_aot_kernel.cuh"
#include <iostream>
#include <sstream>
#include <vector>
#include <string>
#include <hip/hip_runtime_api.h>

TaichiKernel *taichi_kernel = new TaichiKernel();

std::map<uint32_t, TiDataType> taichiTypeMap = {
    {0, TI_DATA_TYPE_I32},
    {1, TI_DATA_TYPE_F32},
    {2, TI_DATA_TYPE_U1},
    {3, TI_DATA_TYPE_U8},
    {4, TI_DATA_TYPE_U16},
    {5, TI_DATA_TYPE_U32},
    {6, TI_DATA_TYPE_U64},
    {7, TI_DATA_TYPE_I8},
    {8, TI_DATA_TYPE_I16},
    {9, TI_DATA_TYPE_I64},
    {10, TI_DATA_TYPE_F16},
    {11, TI_DATA_TYPE_F64}
};

void push_input(const uint32_t type_id, const void* value, uint32_t dim_count, uint32_t elem_count, uint32_t* shape) {
    switch (type_id)
    {
    case 0:
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<int>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 1:
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<float>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 2:
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<bool>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 3:
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<uint8_t>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 4:
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<uint16_t>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 5:
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<uint32_t>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 6:
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<uint64_t>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 7:
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<int8_t>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 8:
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<int16_t>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 9:
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<int64_t>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 10:
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<float>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 11:
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<double>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    default:
        break;
    }
}

void push_output(const uint32_t type_id, const void* value, uint32_t dim_count, uint32_t elem_count, uint32_t* shape) {
    switch (type_id)
    {
    case 0:
        hipMemset(const_cast<void*>(value), 0, elem_count * sizeof(int));
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<int>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 1:
        hipMemset(const_cast<void*>(value), 0, elem_count * sizeof(float));
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<float>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 2:
        hipMemset(const_cast<void*>(value), 0, elem_count * sizeof(bool));
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<bool>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 3:
        hipMemset(const_cast<void*>(value), 0, elem_count * sizeof(uint8_t));
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<uint8_t>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 4:
        hipMemset(const_cast<void*>(value), 0, elem_count * sizeof(uint16_t));
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<uint16_t>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 5:
        hipMemset(const_cast<void*>(value), 0, elem_count * sizeof(uint32_t));
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<uint32_t>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 6:
        hipMemset(const_cast<void*>(value), 0, elem_count * sizeof(uint64_t));
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<uint64_t>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 7:
        hipMemset(const_cast<void*>(value), 0, elem_count * sizeof(int8_t));
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<int8_t>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 8:
        hipMemset(const_cast<void*>(value), 0, elem_count * sizeof(int16_t));
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<int16_t>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 9:
        hipMemset(const_cast<void*>(value), 0, elem_count * sizeof(int64_t));
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<int64_t>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 10:
        hipMemset(const_cast<void*>(value), 0, elem_count * sizeof(float));
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<float>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    case 11:
        hipMemset(const_cast<void*>(value), 0, elem_count * sizeof(double));
        taichi_kernel->kernel->push_arg(createNdArrayFromRawMemory<double>(const_cast<void*>(value), dim_count, elem_count, type_id, shape));
        break;

    default:
        break;
    }
}


TiDataType getTiDataTypeFromMap(uint32_t typeIndex) {
    return taichiTypeMap[typeIndex];
}

OpaqueStruct parseOpaque(const char* opaque, std::size_t opque_len) {
    std::string input(opaque, opque_len);
    std::stringstream ss(input);
    std::string segment;

    OpaqueStruct data;

    // parse in_num and out_num
    if (std::getline(ss, segment, ';')) {
        std::stringstream nums(segment);
        std::string num;
        std::getline(nums, num, ',');
        data.in_num = std::stoul(num);
        std::getline(nums, num, ',');
        data.out_num = std::stoul(num);
    }

    // Helper function to parse a list of uint32_t
    auto parseList = [&ss](std::vector<uint32_t>& vec) {
        std::string segment;
        if (std::getline(ss, segment, ';')) {
            std::stringstream listStream(segment);
            std::string item;
            while (std::getline(listStream, item, ',')) {
                vec.push_back(std::stoul(item));
            }
        }
    };

    // Parse remaining lists
    parseList(data.type_list);
    parseList(data.ndim_list);
    parseList(data.size_list);
    parseList(data.shape_list);

    // Parse kernel_aot_path
    std::getline(ss, data.kernel_aot_path, ';');

    return data;
}
